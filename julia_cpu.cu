/*
Code adapted from book "CUDA by Example: An Introduction to General-Purpose GPU Programming" 

This code computes a visualization of the Julia set.  Two-dimensional "bitmap" data which can be plotted is computed by the function kernel.

The data can be viewed with gnuplot.

The Julia set iteration is:

z= z**2 + C

If it converges, then the initial point z is in the Julia set.

This code is CPU only but will compile with:

nvcc julia_cpu.cu

 
*/


#include <stdio.h>
#include <hip/hip_runtime.h>

#define DIM 1000

__device__ int julia( int x, int y ) { 
    const float scale = 1.5;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);

    float cr=-0.8f;
    float ci=0.156f;

    float ar=jx;
    float ai=jy;
    float artmp;

    int i = 0;
    for (i=0; i<200; i++) {

        artmp = ar;
        ar =(ar*ar-ai*ai) +cr;
        ai = 2.0f*artmp*ai + ci;

        if ( (ar*ar+ai*ai) > 1000)
            return 0;
    }

    return 1;
}

__global__ void  kernel( int *arr_d, int n ){
  

  int x,y;
  x=blockIdx.x * blockDim.x + threadIdx.x;
  y=blockIdx.y * blockDim.y + threadIdx.y;
 
  int offset = x + y * DIM;

  int juliaValue = julia( x, y );
  arr_d[offset] = juliaValue;
        
    
}

int main( void ) {
    int *arr_h;
    int *arr_d;
    int *arr_shadow;
    
    int n = DIM*DIM;
    size_t memsize; 
    memsize = n * sizeof(int); 



    arr_h = (int *)malloc(memsize);
    hipMalloc((void **) &arr_d, memsize);
    arr_shadow = (int *)malloc(memsize);
    
    hipMemcpy(arr_h,arr_d,memsize, hipMemcpyHostToDevice);   
    dim3 gridDef1(DIM,DIM,1);
    dim3 blockDef1(1,1,1); 
    
    FILE *out;
 
    //execute kernel
    kernel<<<gridDef1,blockDef1>>>(arr_d, n); 

    //Retrieve results
    hipMemcpy(arr_shadow, arr_d, memsize, hipMemcpyDeviceToHost); 

    //Ensure synchronization
    hipDeviceSynchronize(); 

    out = fopen( "julia.dat", "w" );
    for (int y=0; y<DIM; y++) {
        for (int x=0; x<DIM; x++) {
            int offset = x + y * DIM;
            if(arr_shadow[offset]==1)
                fprintf(out,"%d %d \n",x,y);  
        } 
    } 
    fclose(out);

}

